#include "hip/hip_runtime.h"
#include "Solver.cuh"

__constant__ int d_numOfNodes;
__constant__ int d_numOfVertices;

__global__ void kernelInitNodes(DeviceData d_origData, DeviceData d_tranData);
__global__ void kernelInitVertices(DeviceData d_origData, DeviceData d_tranData);
__global__ void kernelSolveNodes(DeviceData d_origData, DeviceData d_tranData);
__global__ void kernelSolveVertices(DeviceData d_origData, DeviceData d_tranData);

Solver & Solver::getInstance()
{
	static Solver solver;
	return solver;
}

void Solver::run(HostData & h_origData, HostData & h_tranData)
{
	// allocate device memory
	mallocData(h_origData, h_tranData);

	// copy data from host to device
	copyDataFromHostToDevice(h_origData, h_tranData);

	// execute nonlinear solver with CUDA kernels
	executeKernel(h_origData, h_tranData);

	// copy data from device to host
	copyDataFromDeviceToHost(h_origData, h_tranData);
}

void Solver::mallocData(const HostData & h_origData, const HostData & h_tranData)
{
	hipError_t result;

	// allocate device memory of original data
	result = hipMalloc(&d_origData.nodes, h_origData.nodes.size() * sizeof(Node));
	if (result != hipSuccess) throw std::runtime_error("Failed to allocate device memory");
	result = hipMalloc(&d_origData.vertices, h_origData.vertices.size() * sizeof(Vertex));
	if (result != hipSuccess) throw std::runtime_error("Failed to allocate device memory");

	// allocate device memory of transformed data
	result = hipMalloc(&d_tranData.nodes, h_tranData.nodes.size() * sizeof(Node));
	if (result != hipSuccess) throw std::runtime_error("Failed to allocate device memory");
	result = hipMalloc(&d_tranData.vertices, h_tranData.vertices.size() * sizeof(Vertex));
	if (result != hipSuccess) throw std::runtime_error("Failed to allocate device memory");
}

void Solver::copyDataFromHostToDevice(const HostData & h_origData, const HostData & h_tranData) const
{
	hipError_t result;

	// copy original data from host to device
	result = hipMemcpy(d_origData.nodes, h_origData.nodes.data(),
		h_origData.nodes.size() * sizeof(Node), hipMemcpyHostToDevice);
	if (result != hipSuccess) throw std::runtime_error("Failed to copy data from host to device");
	result = hipMemcpy(d_origData.vertices, h_origData.vertices.data(),
		h_origData.vertices.size() * sizeof(Vertex), hipMemcpyHostToDevice);
	if (result != hipSuccess) throw std::runtime_error("Failed to copy data from host to device");

	// copy transformed data from host to device
	result = hipMemcpy(d_tranData.nodes, h_tranData.nodes.data(),
		h_tranData.nodes.size() * sizeof(Node), hipMemcpyHostToDevice);
	if (result != hipSuccess) throw std::runtime_error("Failed to copy data from host to device");
	result = hipMemcpy(d_tranData.vertices, h_tranData.vertices.data(),
		h_tranData.vertices.size() * sizeof(Vertex), hipMemcpyHostToDevice);
	if (result != hipSuccess) throw std::runtime_error("Failed to copy data from host to device");

	// copy the number of nodes
	const int h_numOfNodes = h_origData.nodes.size();
	hipMemcpyToSymbol(HIP_SYMBOL(d_numOfNodes), &h_numOfNodes, sizeof(int));

	// copy the number of vertices
	const int h_numOfVertices = h_origData.vertices.size();
	hipMemcpyToSymbol(HIP_SYMBOL(d_numOfVertices), &h_numOfVertices, sizeof(int));
}

void Solver::copyDataFromDeviceToHost(HostData & h_origData, HostData & h_tranData) const
{
	hipError_t result;

	// copy original data from device to host
	result = hipMemcpy(h_origData.nodes.data(), d_origData.nodes,
		h_origData.nodes.size() * sizeof(Node), hipMemcpyDeviceToHost);
	if (result != hipSuccess) throw std::runtime_error("Failed to copy data from device to host");
	result = hipMemcpy(h_origData.vertices.data(), d_origData.vertices,
		h_origData.vertices.size() * sizeof(Vertex), hipMemcpyDeviceToHost);
	if (result != hipSuccess) throw std::runtime_error("Failed to copy data from device to host");

	// copy transformed data from device to host
	result = hipMemcpy(h_tranData.nodes.data(), d_tranData.nodes,
		h_tranData.nodes.size() * sizeof(Node), hipMemcpyDeviceToHost);
	if (result != hipSuccess) throw std::runtime_error("Failed to copy data from device to host");
	result = hipMemcpy(h_tranData.vertices.data(), d_tranData.vertices,
		h_tranData.vertices.size() * sizeof(Vertex), hipMemcpyDeviceToHost);
	if (result != hipSuccess) throw std::runtime_error("Failed to copy data from device to host");
}

void Solver::executeKernel(const HostData & h_origData, const HostData & h_tranData) const
{
	// get the number of nodes and vertices
	const int numOfNodes = h_origData.nodes.size();
	const int numOfVertices = h_origData.vertices.size();

	// compute dimension of nodes and vertices for CUDA kernels
	const int dimOfNodes = (numOfNodes + Config::threadPerBlock - 1) / Config::threadPerBlock;
	const int dimOfVertices = (numOfVertices + Config::threadPerBlock - 1) / Config::threadPerBlock;

	// execute initializing kernels
	kernelInitNodes << <dimOfNodes, Config::threadPerBlock >> > (d_origData, d_tranData);
	kernelInitVertices << <dimOfVertices, Config::threadPerBlock >> > (d_origData, d_tranData);

	// execute nonlinear solving kernels with iterations
	for (int i = 0; i < Config::numOfIterations; i++)
	{
		kernelSolveNodes << <dimOfNodes, Config::threadPerBlock >> > (d_origData, d_tranData);
		kernelSolveVertices << <dimOfVertices, Config::threadPerBlock >> > (d_origData, d_tranData);
	}
}

__global__ void kernelInitNodes(DeviceData d_origData, DeviceData d_tranData)
{
	const int index = threadIdx.x + blockIdx.x * blockDim.x;

	// set initial transformed poses to original poses
	d_tranData.nodes[index].pose = d_origData.nodes[index].pose;
}

__global__ void kernelInitVertices(DeviceData d_origData, DeviceData d_tranData)
{
	const int index = threadIdx.x + blockIdx.x * blockDim.x;

	// do nothing
}

__global__ void kernelSolveNodes(DeviceData d_origData, DeviceData d_tranData)
{
	const int index = threadIdx.x + blockIdx.x * blockDim.x;

	// TODO: do nonlinear solve for each node
}

__global__ void kernelSolveVertices(DeviceData d_origData, DeviceData d_tranData)
{
	const int index = threadIdx.x + blockIdx.x * blockDim.x;

	// TODO: do nonlinear solve for each vertex
}
